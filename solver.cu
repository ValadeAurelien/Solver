#ifndef SOLVER_H
#define SOLVER_H

#ifdef __HIPCC__
#define CUDA_CALLABLE_MEMBER __host__ __device__
#else
#define CUDA_CALLABLE_MEMBER
#endif 
#include <iostream>

using namespace std;


/*
class TemplateBaseClass_t
{
  public:
    CUDA_CALLABLE_MEMBER 
      virtual void operator=(TemplateBaseClass_t &_TemplateBaseClass);
    CUDA_CALLABLE_MEMBER 
      virtual TemplateBaseClass_t operator+(const TemplateBaseClass_t &_TemplateBaseClass) const ;
    CUDA_CALLABLE_MEMBER 
      virtual TemplateBaseClass_t operator-(const TemplateBaseClass_t &_TemplateBaseClass) const ;
    CUDA_CALLABLE_MEMBER 
      virtual TemplateBaseClass_t operator*(const float_tt &d) const ;
    CUDA_CALLABLE_MEMBER 
      virtual TemplateBaseClass_t operator/(const float_tt &d) const ;
    CUDA_CALLABLE_MEMBER 
      virtual TemplateBaseClass_t norm() const ;
};

template <typename BaseClass_t>
class TemplateCallableClass_t
{
  public:
    CUDA_CALLABLE_MEMBER 
      virtual BaseClass_t operator()(const BaseClass_t& _BaseClass) const;
};
*/

template<typename float_tt>
struct err_n_dt_t
{
  CUDA_CALLABLE_MEMBER
    void operator=(const err_n_dt_t<float_tt>& _end) { err = _end.err; nbs = _end.nbs; dt = _end.dt; } 
  CUDA_CALLABLE_MEMBER
    void set(float_tt _err, unsigned int _nbs, float_tt _dt){ err = _err; nbs = _nbs; dt = _dt; }
  float_tt err, dt;
  unsigned int nbs;
};

struct RKF_t
{
  RKF_t(){}
  double a21 = 1./4,
         a31 = 3./32, a32 = 9./32,
         a41 = 1932./2197, a42 = -7200./2197, a43 = 7296./2197,
         a51 = 439./216, a52 = -8., a53 = 3680./513, a54 = -845./4104,
         a61 = -8./27, a62 = 2., a63 = -3544./2565, a64 = 1859./4104, a65 = -11./40,
         b1 = 16./135, b2 = 0., b3 = 6656./12825, b4 = 28561./56430, b5 = -9./50, b6 = 2./55,
         bb1 = 25./216, bb2 = 0., bb3 = 1408./2565, bb4 = 2197./4104, bb5 = -1./5, bb6 = 0.; 
};  

template <typename CallableClass_t, typename BaseClass_t, typename float_tt>
class RK45Solver_t
{
  private :
    float_tt dt_min;
    unsigned int nb_steps_max;
    const RKF_t RKF;
    float_tt dt, err, tol;
    err_n_dt_t<float_tt> err_n_dt;
    BaseClass_t BaseClass_out_deg;
    CallableClass_t Callable;

    CUDA_CALLABLE_MEMBER
      float_tt one_step(const BaseClass_t& _BaseClass_in, BaseClass_t& _BaseClass_out);

  public:
    CUDA_CALLABLE_MEMBER
      RK45Solver_t(CallableClass_t& _Callable, float_tt _dt_min, unsigned int _nb_steps_max);
    CUDA_CALLABLE_MEMBER
      CallableClass_t& get_CallableClass() const;
    CUDA_CALLABLE_MEMBER 
      bool operator()(float_tt _dt, float_tt _tol, const BaseClass_t& _BaseClass_in, BaseClass_t& _BaseClass_out);
    CUDA_CALLABLE_MEMBER
      const err_n_dt_t<float_tt>& get_err_n_dt() const;
};


template <typename CallableClass_t, typename BaseClass_t, typename float_tt>
CUDA_CALLABLE_MEMBER
RK45Solver_t <CallableClass_t, BaseClass_t, float_tt>::RK45Solver_t(CallableClass_t& _Callable, float_tt _dt_min, unsigned int _nb_steps_max) : 
  Callable(_Callable), dt_min (_dt_min), nb_steps_max (_nb_steps_max) {}
  
template <typename CallableClass_t, typename BaseClass_t, typename float_tt>
CUDA_CALLABLE_MEMBER
float_tt RK45Solver_t<CallableClass_t, BaseClass_t, float_tt>::one_step(const BaseClass_t& _BaseClass_in, BaseClass_t& _BaseClass_out)
{
  const BaseClass_t &X = _BaseClass_in;
  BaseClass_t &Y = _BaseClass_out,
              &Ydeg = BaseClass_out_deg,
              K1, K2, K3, K4, K5, K6,
              K1int, K2int, K3int, K4int, K5int, K6int,
              dX;

  K1 = Callable(X);

  K1int = K1*RKF.a21*dt;
  K2 = Callable(K1int+X);

  K1int = K1*RKF.a31*dt;
  K2int = K2*RKF.a32*dt;
  K3 = Callable(K2int+K1int+X);
  
  K1int = K1*RKF.a41*dt;
  K2int = K2*RKF.a42*dt;
  K3int = K3*RKF.a43*dt;
  K4 = Callable(K3int+K2int+K1int+X);

  K1int = K1*RKF.a51*dt;
  K2int = K2*RKF.a52*dt;
  K3int = K3*RKF.a53*dt;
  K4int = K4*RKF.a54*dt;
  K5 = Callable(K4int+K3int+K2int+K1int+X);

  K1int = K1*RKF.a61*dt;
  K2int = K2*RKF.a62*dt;
  K3int = K3*RKF.a63*dt;
  K4int = K4*RKF.a64*dt;
  K5int = K5*RKF.a65*dt;
  K6 = Callable(K4int+K3int+K2int+K1int+X);
  
  dX = (K1*RKF.b1 + K2*RKF.b2 + K3*RKF.b3 + K4*RKF.b4 + K5*RKF.b5 + K6*RKF.b6)*dt;
  Y = X + dX;

  dX = (K1*(RKF.b1-RKF.bb1) + K2*(RKF.b2-RKF.bb2) + K3*(RKF.b3-RKF.bb3) + K4*(RKF.b4-RKF.bb4) + K5*(RKF.b5-RKF.bb5) + K6*(RKF.b6-RKF.bb6))*dt;
  Ydeg = X + dX;

  return (Y-Ydeg).norm();
}

template <typename CallableClass_t, typename BaseClass_t, typename float_tt>
CUDA_CALLABLE_MEMBER
bool RK45Solver_t<CallableClass_t, BaseClass_t, float_tt>::operator()(float_tt _dt, float_tt _tol, const BaseClass_t& _BaseClass_in, BaseClass_t& _BaseClass_out)
{ 
  dt = _dt; tol = _tol;
  int nb_steps=0;
  for(float_tt t=0; t<_dt; t+=dt)
  {
    err = one_step(_BaseClass_in, _BaseClass_out);
    while (err>tol) 
    {
      dt = .9*powf(tol/err, 1./3);
      nb_steps++;
      if (dt<dt_min || nb_steps>nb_steps_max)
      {
        err_n_dt.set(err, nb_steps, dt);
        return false;
      }
      err = one_step(_BaseClass_in, _BaseClass_out);
    } 
  }
  err_n_dt.set(err, nb_steps, dt);
  return true;
}

template <typename CallableClass_t, typename BaseClass_t, typename float_tt>
CUDA_CALLABLE_MEMBER
const err_n_dt_t<float_tt>& RK45Solver_t<CallableClass_t, BaseClass_t, float_tt>::get_err_n_dt() const { return err_n_dt; }

#endif
